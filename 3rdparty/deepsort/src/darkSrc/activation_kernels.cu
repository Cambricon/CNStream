#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "activations.h"
#include "hip/hip_runtime.h"
}


__device__ float lhtan_activate_kernel(float x)
{
    if(x < 0) return .001*x;
    if(x > 1) return .001*(x-1) + 1;
    return x;
}
__device__ float lhtan_gradient_kernel(float x)
{
    if(x > 0 && x < 1) return 1;
    return .001;
}

__device__ float hardtan_activate_kernel(float x)
{
    if (x < -1) return -1;
    if (x > 1) return 1;
    return x;
}
__device__ float linear_activate_kernel(float x){return x;}
__device__ float logistic_activate_kernel(float x){return 1./(1. + exp(-x));}
__device__ float loggy_activate_kernel(float x){return 2./(1. + exp(-x)) - 1;}
__device__ float relu_activate_kernel(float x){return x*(x>0);}
__device__ float elu_activate_kernel(float x){return (x >= 0)*x + (x < 0)*(exp(x)-1);}
__device__ float relie_activate_kernel(float x){return (x>0) ? x : .01*x;}
__device__ float ramp_activate_kernel(float x){return x*(x>0)+.1*x;}
__device__ float leaky_activate_kernel(float x){return (x>0) ? x : .1*x;}
__device__ float tanh_activate_kernel(float x){return (2/(1 + exp(-2*x)) - 1);}
__device__ float plse_activate_kernel(float x)
{
    if(x < -4) return .01 * (x + 4);
    if(x > 4)  return .01 * (x - 4) + 1;
    return .125*x + .5;
}
__device__ float stair_activate_kernel(float x)
{
    int n = floor(x);
    if (n%2 == 0) return floor(x/2.);
    else return (x - n) + floor(x/2.);
}
 

__device__ float hardtan_gradient_kernel(float x)
{
    if (x > -1 && x < 1) return 1;
    return 0;
}
__device__ float linear_gradient_kernel(float x){return 1;}
__device__ float logistic_gradient_kernel(float x){return (1-x)*x;}
__device__ float loggy_gradient_kernel(float x)
{
    float y = (x+1.)/2.;
    return 2*(1-y)*y;
}
__device__ float relu_gradient_kernel(float x){return (x>0);}
__device__ float elu_gradient_kernel(float x){return (x >= 0) + (x < 0)*(x + 1);}
__device__ float relie_gradient_kernel(float x){return (x>0) ? 1 : .01;}
__device__ float ramp_gradient_kernel(float x){return (x>0)+.1;}
__device__ float leaky_gradient_kernel(float x){return (x>0) ? 1 : .1;}
__device__ float tanh_gradient_kernel(float x){return 1-x*x;}
__device__ float plse_gradient_kernel(float x){return (x < 0 || x > 1) ? .01 : .125;}
__device__ float stair_gradient_kernel(float x)
{
    if (floor(x) == x) return 0;
    return 1;
}

__device__ float activate_kernel(float x, ACTIVATION a)
{
    switch(a){
        case LINEAR:
            return linear_activate_kernel(x);
        case LOGISTIC:
            return logistic_activate_kernel(x);
        case LOGGY:
            return loggy_activate_kernel(x);
        case RELU:
            return relu_activate_kernel(x);
        case ELU:
            return elu_activate_kernel(x);
        case RELIE:
            return relie_activate_kernel(x);
        case RAMP:
            return ramp_activate_kernel(x);
        case LEAKY:
            return leaky_activate_kernel(x);
        case TANH:
            return tanh_activate_kernel(x);
        case PLSE:
            return plse_activate_kernel(x);
        case STAIR:
            return stair_activate_kernel(x);
        case HARDTAN:
            return hardtan_activate_kernel(x);
        case LHTAN:
            return lhtan_activate_kernel(x);
    }
    return 0;
}

__device__ float gradient_kernel(float x, ACTIVATION a)
{
    switch(a){
        case LINEAR:
            return linear_gradient_kernel(x);
        case LOGISTIC:
            return logistic_gradient_kernel(x);
        case LOGGY:
            return loggy_gradient_kernel(x);
        case RELU:
            return relu_gradient_kernel(x);
        case ELU:
            return elu_gradient_kernel(x);
        case RELIE:
            return relie_gradient_kernel(x);
        case RAMP:
            return ramp_gradient_kernel(x);
        case LEAKY:
            return leaky_gradient_kernel(x);
        case TANH:
            return tanh_gradient_kernel(x);
        case PLSE:
            return plse_gradient_kernel(x);
        case STAIR:
            return stair_gradient_kernel(x);
        case HARDTAN:
            return hardtan_gradient_kernel(x);
        case LHTAN:
            return lhtan_gradient_kernel(x);
    }
    return 0;
}

__global__ void activate_array_kernel(float *x, int n, ACTIVATION a)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n) x[i] = activate_kernel(x[i], a);
}

__global__ void gradient_array_kernel(float *x, int n, ACTIVATION a, float *delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n) delta[i] *= gradient_kernel(x[i], a);
}

extern "C" void activate_array_ongpu(float *x, int n, ACTIVATION a) 
{
    activate_array_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, a);
    check_error(hipPeekAtLastError());
}

extern "C" void gradient_array_ongpu(float *x, int n, ACTIVATION a, float *delta) 
{
    gradient_array_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, a, delta);
    check_error(hipPeekAtLastError());
}
