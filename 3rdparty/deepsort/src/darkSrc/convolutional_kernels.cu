#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")  
#endif

extern "C" {
#include "convolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    binary[i] = (x[i] >= 0) ? 1 : -1;
}

void binarize_gpu(float *x, int n, float *binary)
{
    binarize_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, binary);
    check_error(hipPeekAtLastError());
}

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < n; ++i){
        mean += abs(input[i*size + s]);
    }
    mean = mean / n;
    for(i = 0; i < n; ++i){
        binary[i*size + s] = (input[i*size + s] > 0) ? mean : -mean;
    }
}

void binarize_input_gpu(float *input, int n, int size, float *binary)
{
    binarize_input_kernel<<<cuda_gridsize(size), BLOCK>>>(input, n, size, binary);
    check_error(hipPeekAtLastError());
}


__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < size; ++i){
        mean += abs(weights[f*size + i]);
    }
    mean = mean / size;
    for(i = 0; i < size; ++i){
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
        //binary[f*size + i] = weights[f*size + i];
    }
}

void binarize_weights_gpu(float *weights, int n, int size, float *binary)
{
    binarize_weights_kernel<<<cuda_gridsize(n), BLOCK>>>(weights, n, size, binary);
    check_error(hipPeekAtLastError());
}

void forward_convolutional_layer_gpu(convolutional_layer l, network_state state)
{
    fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);
    if(l.binary){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
    }

    if(l.xnor){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
        binarize_gpu(state.input, l.c*l.h*l.w*l.batch, l.binary_input_gpu);
        state.input = l.binary_input_gpu;
    }

#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionForward(cudnn_handle(),
                &one,
                l.srcTensorDesc,
                state.input,
                l.weightDesc,
                l.weights_gpu,
                l.convDesc,
                l.fw_algo,
                state.workspace,
                l.workspace_size,
                &one,
                l.dstTensorDesc,
                l.output_gpu);

#else
    int i;
    int m = l.n;
    int k = l.size*l.size*l.c;
    int n = l.out_w*l.out_h;
    for(i = 0; i < l.batch; ++i){
        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.workspace);
        float * a = l.weights_gpu;
        float * b = state.workspace;
        float * c = l.output_gpu;
        gemm_ongpu(0,0,m,n,k,1.,a,k,b,n,1.,c+i*m*n,n);
    }
#endif

    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, state);
    }
    add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);

    activate_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation);
    //if(l.dot > 0) dot_error_gpu(l);
    if(l.binary || l.xnor) swap_binary(&l);
	//hipDeviceSynchronize();	// for correct profiling of performance
}

void backward_convolutional_layer_gpu(convolutional_layer l, network_state state)
{
    gradient_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);

    backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);

    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, state);
        //axpy_ongpu(l.outputs*l.batch, -state.net.decay, l.x_gpu, 1, l.delta_gpu, 1);
    } else {
        //axpy_ongpu(l.outputs*l.batch, -state.net.decay, l.output_gpu, 1, l.delta_gpu, 1);
    }
    float *original_input = state.input;

    if(l.xnor) state.input = l.binary_input_gpu;
#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionBackwardFilter(cudnn_handle(),
            &one,
            l.srcTensorDesc,
            state.input,
            l.ddstTensorDesc,
            l.delta_gpu,
            l.convDesc,
            l.bf_algo,
            state.workspace,
            l.workspace_size,
            &one,
            l.dweightDesc,
            l.weight_updates_gpu);

    if(state.delta){
        if(l.binary || l.xnor) swap_binary(&l);
        hipdnnConvolutionBackwardData(cudnn_handle(),
                &one,
                l.weightDesc,
                l.weights_gpu,
                l.ddstTensorDesc,
                l.delta_gpu,
                l.convDesc,
                l.bd_algo,
                state.workspace,
                l.workspace_size,
                &one,
                l.dsrcTensorDesc,
                state.delta);
        if(l.binary || l.xnor) swap_binary(&l);
        if(l.xnor) gradient_array_ongpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, state.delta);
    }

#else
    int m = l.n;
    int n = l.size*l.size*l.c;
    int k = l.out_w*l.out_h;

    int i;
    for(i = 0; i < l.batch; ++i){
        float * a = l.delta_gpu;
        float * b = state.workspace;
        float * c = l.weight_updates_gpu;

        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.workspace);
        gemm_ongpu(0,1,m,n,k,1,a + i*m*k,k,b,k,1,c,n);

        if(state.delta){
            if(l.binary || l.xnor) swap_binary(&l);
            float * a = l.weights_gpu;
            float * b = l.delta_gpu;
            float * c = state.workspace;

            gemm_ongpu(1,0,n,k,m,1,a,n,b + i*k*m,k,0,c,k);

            col2im_ongpu(state.workspace, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.delta + i*l.c*l.h*l.w);
            if(l.binary || l.xnor) {
                swap_binary(&l);
            }
            if(l.xnor) gradient_array_ongpu(original_input + i*l.c*l.h*l.w, l.c*l.h*l.w, HARDTAN, state.delta + i*l.c*l.h*l.w);
        }
    }
#endif
}

void pull_convolutional_layer(convolutional_layer layer)
{
    cuda_pull_array(layer.weights_gpu, layer.weights, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.weight_updates_gpu, layer.weight_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_pull_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_pull_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_pull_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
    if (layer.adam){
        cuda_pull_array(layer.m_gpu, layer.m, layer.c*layer.n*layer.size*layer.size);
        cuda_pull_array(layer.v_gpu, layer.v, layer.c*layer.n*layer.size*layer.size);
    }
}

void push_convolutional_layer(convolutional_layer layer)
{
    cuda_push_array(layer.weights_gpu, layer.weights, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.weight_updates_gpu, layer.weight_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_push_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_push_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_push_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
    if (layer.adam){
        cuda_push_array(layer.m_gpu, layer.m, layer.c*layer.n*layer.size*layer.size);
        cuda_push_array(layer.v_gpu, layer.v, layer.c*layer.n*layer.size*layer.size);
    }
}

void update_convolutional_layer_gpu(convolutional_layer layer, int batch, float learning_rate, float momentum, float decay)
{
    int size = layer.size*layer.size*layer.c*layer.n;
    axpy_ongpu(layer.n, learning_rate/batch, layer.bias_updates_gpu, 1, layer.biases_gpu, 1);
    scal_ongpu(layer.n, momentum, layer.bias_updates_gpu, 1);

    if(layer.scales_gpu){
        axpy_ongpu(layer.n, learning_rate/batch, layer.scale_updates_gpu, 1, layer.scales_gpu, 1);
        scal_ongpu(layer.n, momentum, layer.scale_updates_gpu, 1);
    }

    if(layer.adam){
        scal_ongpu(size, layer.B1, layer.m_gpu, 1);
        scal_ongpu(size, layer.B2, layer.v_gpu, 1);

        axpy_ongpu(size, -decay*batch, layer.weights_gpu, 1, layer.weight_updates_gpu, 1);

        axpy_ongpu(size, -(1-layer.B1), layer.weight_updates_gpu, 1, layer.m_gpu, 1);
        mul_ongpu(size, layer.weight_updates_gpu, 1, layer.weight_updates_gpu, 1);
        axpy_ongpu(size, (1-layer.B2), layer.weight_updates_gpu, 1, layer.v_gpu, 1);

        adam_gpu(size, layer.weights_gpu, layer.m_gpu, layer.v_gpu, layer.B1, layer.B2, learning_rate/batch, layer.eps, layer.t+1);
        fill_ongpu(size, 0, layer.weight_updates_gpu, 1);
    }else{
        axpy_ongpu(size, -decay*batch, layer.weights_gpu, 1, layer.weight_updates_gpu, 1);
        axpy_ongpu(size, learning_rate/batch, layer.weight_updates_gpu, 1, layer.weights_gpu, 1);
        scal_ongpu(size, momentum, layer.weight_updates_gpu, 1);
    }
}


